#include "hip/hip_runtime.h"

#include "CUDASleep.h"

__global__ void nanosleepX(unsigned nanos) {
  __nanosleep(nanos);
}

void gpu_nsleep(unsigned nanos, hipStream_t stream) {
  nanosleepX<<<1,1,0,stream>>>(nanos);
}
